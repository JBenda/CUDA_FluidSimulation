#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <atomic>
#include <thread>
#include <chrono>
#include <intrin.h>

#include "d:\Dokumente\OVGU\GPU\cudaSample\solution\src\cuda_util.h"
std::atomic_bool copyDevDesFin = false;
int main()
{

    return 0;
}

void safeFrame(int num, float* picture, float* dev_picture, const int2 res)	//very slow
{
	hipError_t cudaStatus = hipMemcpy(picture, dev_picture, res.x * res.y * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "copy frame from device failed!" << std::endl;
	}
	copyDevDesFin = false;
	FILE *output;
	std::stringstream ss;
	ss << "frame" << num << ".pnm";
	output = fopen(ss.str().c_str(), "w");
	ss.str("");
	ss << "P5 " << res.x << ' ' << res.y << " 255 ";
	fprintf(output, ss.str().c_str());
	for (int i = 0; i < res.x*res.y; ++i)
	{
		if (picture[i] < 0.f)
		{
			std::cerr << "ERROR" << std::endl;
			return;
		}
		fprintf(output, "%c", picture[i] == 0 ? (int)0 : (int)255);
	}
	fclose(output);
}

hipError_t fluidSimulation(const int2 res, const int n, const float diff, const float dt, const int frames, float* pos_x, float* pos_y)
{
	const int pixel = res.x * res.y;
	float *dev_vel_x;
	float *dev_vel_y;
	float *dev_pos_x;
	float *dev_pos_y;
	float *dev_dvl_x;
	float *dev_dvl_y;
	uint8_t *dev_pic;

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (0 == deviceCount) {
		std::cerr << "No CUDA device found." << std::endl;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDeviceProps(devProp);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
	}
	hipMalloc((void**)dev_vel_x, n * sizeof(float));
	hipMalloc((void**)dev_vel_y, n * sizeof(float));
	hipMalloc((void**)dev_pos_x, n * sizeof(float));
	hipMalloc((void**)dev_pos_y, n * sizeof(float));
	hipMalloc((void**)dev_dvl_x, n * sizeof(float));
	hipMalloc((void**)dev_dvl_y, n * sizeof(float));
	hipMalloc((void**)dev_pic, pixel * sizeof(uint8_t));

	hipMemset(dev_vel_x, 0, n * sizeof(float));
	hipMemset(dev_vel_y, 0, n * sizeof(float));
	hipMemset(dev_dvl_x, 0, n * sizeof(float));
	hipMemset(dev_dvl_y, 0, n * sizeof(float));
	hipMemset(dev_pic, 0, n * sizeof(float));

	hipMemcpy(dev_pos_x, pos_x, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_pos_y, pos_y, n * sizeof(float), hipMemcpyHostToDevice);

	const int MAX_THREADS_PER_BLOCK = 1024;	//per dim
	unsigned int blocks, threadsPerBlock;
	threadsPerBlock = std::min(n, MAX_THREADS_PER_BLOCK);
	blocks = n / MAX_THREADS_PER_BLOCK;
	if (n % MAX_THREADS_PER_BLOCK != 0)
		blocks++;
	std::cout << "need " << blocks << " per dim blocks with max " << threadsPerBlock << "per block per dim" << std::endl;
	float* picture = (float*)malloc(pixel * sizeof(float));
	dim3 blockSize = dim3(blocks, blocks);
	dim3 threadSize = dim3(threadsPerBlock, threadsPerBlock);


	std::thread safePicThread;
	const int STEPS_BETWEEN_FRAMES = 30;
	float *dev_p, *dev_diff;		//field to save vel diff and presuare temp
	for (size_t frame = 0; frame <= frames * STEPS_BETWEEN_FRAMES; ++frame)
	{
		//std::cout << "strat " << frame << std::endl;
		if (frame % STEPS_BETWEEN_FRAMES == 0)
		{
			if (safePicThread.joinable())
			{
				safePicThread.join();
				copyDevDesFin = true;
				safePicThread = std::thread(safeFrame, frame, picture, dev_pic, res);
				while (copyDevDesFin);
			}
			else
				__debugbreak();
		}



		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "diffuseKernel failed!" << std::endl;
		}
		cudaStatus = hipDeviceSynchronize();
	}
	safePicThread.join();
	hipFree(dev_vel_x);
	hipFree(dev_vel_y);
	hipFree(dev_pos_x);
	hipFree(dev_pos_y);
	hipFree(dev_dvl_x);
	hipFree(dev_dvl_y);
	hipFree(dev_pic);

	return cudaStatus;
}