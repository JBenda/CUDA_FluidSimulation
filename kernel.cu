#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <atomic>
#include <thread>
#include <chrono>
#include <intrin.h>

#include "d:\Dokumente\OVGU\GPU\cudaSample\solution\src\cuda_util.h"
std::atomic_bool copyDevDesFin = false;
#define N 100
#define AMOUNT_SM 20
__global__ void calculateMovmentKernel(unsigned int *eInG, uint4* areal, const float H, const float dt, const float visc, const float d, const float g, float *pos_x, float *pos_y, float *vel_x, float *vel_y, float *posN_x, float *posN_y, float *velN_x, float *velN_y);
__global__ void sumbissionKernel(const int2 res, const int2 fields, uint4 *dev_areal, uint32_t *dev_eInG, const float H, const float dt, const float visc, const float d, const float g, float *pos_x, float *pos_y, float *vel_x, float *vel_y, float *posN_x, float *posN_y, float *velN_x, float *velN_y)
{
	const int h = h + ((int)h) < h ? 1 : 0;
	const int2 size = { res.x / fields.x, res.y / fields.y };
	__shared__ uint32_t group[N];			//array partikel -> group // 0x00 & group = gruppe 1
	__shared__ uint4 areal[AMOUNT_SM];		//(x,y) top left corner, w = width, z = height
	__shared__ uint32_t eInG[AMOUNT_SM];//particel / group
	if (threadIdx.x < AMOUNT_SM)
	{
		eInG[threadIdx.x] = 0;
		if (threadIdx.x % fields.x == 0)	//left wall
		{
			areal[threadIdx.x].x = 0;
			areal[threadIdx.x].w = size.x + h;
		}
		else if (threadIdx.x % fields.x == fields.x - 1)
		{
			areal[threadIdx.x].x = res.x - areal[threadIdx.x - 1].x - areal[threadIdx.x - 1].w + h;
			areal[threadIdx.x].w = res.x - areal[threadIdx.x].x;
		}
		else
		{
			areal[threadIdx.x].x = areal[threadIdx.x].x + areal[threadIdx.x].w - h;
			areal[threadIdx.x].w = size.x + h + h;
		}

		if (threadIdx.y / fields.x == 0)	//left wall
		{
			areal[threadIdx.x].y = 0;
			areal[threadIdx.x].z = size.y + h;
		}
		else if (threadIdx.x / fields.x == fields.y - 1)
		{
			areal[threadIdx.x].y = res.y - areal[threadIdx.x - 1].y - areal[threadIdx.x - 1].z + h;
			areal[threadIdx.x].z = res.y - areal[threadIdx.x].y;
		}
		else
		{
			areal[threadIdx.x].y = areal[threadIdx.x].x + areal[threadIdx.x].z - h;
			areal[threadIdx.x].z = size.y + h + h;
		}
	}
	__syncthreads();

	for (int i = 0; i * blockDim.x + threadIdx.x < N; ++i)
	{
		unsigned int x = pos_x[i * blockDim.x + threadIdx.x] / (size.x + h);	//x = x cordinate von min group
		unsigned int y = pos_y[i * blockDim.x + threadIdx.x] / (size.y + h);
		unsigned int gNr = x + y * fields.x;
		group[i * blockDim.x + threadIdx.x] = 0x00000000 | (0x00000001 << gNr);
		atomicAdd(eInG + gNr, 1);
		if(pos_x[i * blockDim.x + threadIdx.x] - x * (size.x + h) >= size.x && x < fields.x - 1)	//im geteilten bereich zwischen zwei Boxen horizontal
		{
			group[i * blockDim.x + threadIdx.x] |= (0x00000001 << (gNr + 1));
			atomicAdd(eInG + gNr + 1, 1);
			if (pos_y[i * blockDim.x + threadIdx.x] - y * (size.y + h) >= size.y && y < fields.y - 1)	//im geteilten breich zwischen vier Boxen 
			{
				group[i * blockDim.x + threadIdx.x] |= (0x00000001 << (gNr + fields.x));
				group[i * blockDim.x + threadIdx.x] |= (0x00000001 << (gNr + fields.x + 1));
				eInG[gNr + fields.x] ++;
				atomicAdd(eInG + gNr + fields.x + 1, 1);
			}
		}
		else if (pos_y[i * blockDim.x + threadIdx.x] - y * (size.y + h) >= size.y && y < fields.y - 1)	//im geteilten breich zwischen zewi Boxen horizontal
		{
			group[i * blockDim.x + threadIdx.x] |= (0x00000001 << (gNr + fields.x));
			atomicAdd(eInG + gNr + fields.x, 1);
		}
	}
	__syncthreads();

	if (threadIdx.x < AMOUNT_SM)
	{
		dev_areal[threadIdx.x] = areal[threadIdx.x];
		dev_eInG[threadIdx.x] = eInG[threadIdx.x];
	}
	unsigned int maxE = 0;
	if (threadIdx.x == 0)
	{
		for (int i = 0; i < AMOUNT_SM; ++i)
			if (eInG[i] > maxE)
				maxE = eInG[i] > maxE;
	}
	__syncthreads();
	dim3 blocks(fields.x, fields.y);
	calculateMovmentKernel << <1024, blocks, 5 * maxE * sizeof(float) + maxE * sizeof(uint32_t)>> > (dev_eInG, dev_areal, H, dt, visc, d, g, pos_x, pos_y, vel_x, vel_y, posN_x, posN_y, velN_x, velN_y);
}
__global__ void calculateMovmentKernel(unsigned int *eInG, uint4* areal, const float H, const float dt, const float visc, const float d, const float g, float *pos_x, float *pos_y, float *vel_x, float *vel_y, float *posN_x, float *posN_y, float *velN_x, float *velN_y)
{
	extern __shared__ float *shared;
	const unsigned int eInA= eInG[blockIdx.x];
	float *pres;
	float *s_pos_x = shared + eInA;
	float *s_pos_y = shared + 2 * eInA;
	float *s_vel_x = shared + 3 * eInA;
	float *s_vel_y = shared + 4 * eInA;
	uint32_t *id = (uint32_t*)(shared + 5 * eInA);
	const float min = H / 30.f;
	const float max = 100.f;
	uint4 a = areal[blockIdx.x];
	__shared__ unsigned int pos;
	if (threadIdx.x == 0)
		pos = -1;
	for (unsigned int i = 0; i * blockDim.x + threadIdx.x < N; ++i)
	{
		if(pos_x[i * blockDim.x + threadIdx.x] > a.x && pos_x[i * blockDim.x + threadIdx.x] < (a.x + a.w))
			if (pos_y[i * blockDim.x + threadIdx.x] > a.y && pos_y[i * blockDim.x + threadIdx.x] < (a.y + a.z))
			{
				unsigned int p = atomicAdd(&pos, 1);
				s_pos_x[p] = pos_x[i * blockDim.x + threadIdx.x];
				s_pos_y[p] = pos_y[i * blockDim.x + threadIdx.x];
				s_vel_x[p] = vel_x[i * blockDim.x + threadIdx.x];
				s_vel_y[p] = vel_y[i * blockDim.x + threadIdx.x];
				id[p] = i * blockDim.x + threadIdx.x;
			}
	}
	__syncthreads();
	float dxSq;
	float2 dx;
	for (unsigned int i = 0; i * blockDim.x + threadIdx.x <= pos; ++i)
	{
		pres[i * blockDim.x + threadIdx.x] = 0.f;
		for (unsigned int j = 0; j <= pos; ++j)
		{
			dx.x = s_pos_x[j] - s_pos_x[i * blockDim.x + threadIdx.x];
			dx.y = s_pos_y[j] - s_pos_y[i * blockDim.x + threadIdx.x];
			dxSq = dx.x*dx.x + dx.y*dx.y;
			if (dxSq < H*H)
			{
				if (dxSq < min*min)
					pres[i * blockDim.x + threadIdx.x] += 1.f / (min*min);
				else
					pres[i * blockDim.x + threadIdx.x] += 1.f / dxSq;
			}
		}
	}
	__syncthreads();
	float2 dv = { 0.f, g };
	float absDx;
	for (unsigned int i = 0; i * blockDim.x + threadIdx.x <= pos; ++i)
	{
		int k = i * blockDim.x + threadIdx.x;
		if (s_pos_x[k] > a.x + a.w - H && ! (blockIdx.x == gridDim.x - 1)
			|| s_pos_y[k] > a.y + a.z - H && ! (blockIdx.y == gridDim.y - 1))	//if in border area and not end of screen
				continue;
		for (unsigned int j = 0; j <= pos; ++j)
		{
			dx.x = s_pos_x[j] - s_pos_x[k];
			dx.y = s_pos_y[j] - s_pos_y[k];
			dxSq = dx.x*dx.x + dx.y*dx.y;
			if (dxSq < min*min)
			{
				//calculate dVel from pressuar
				dv.x -= dx.x * (pres[j] + pres[k]) / min * d;
				dv.y -= dx.y * (pres[j] + pres[k]) / min * d;
			}
			else
			{ 
				absDx = std::sqrt(dxSq);
				//calculate dVel from pressuar
				dv.x -= dx.x * (pres[j] + pres[k]) / absDx * d;
				dv.y -= dx.y * (pres[j] + pres[k]) / absDx * d;
				//alculate dVel from visc
				dx = { -dx.y, dx.x };		//rotate 90�
				float v1_x = dx.x * (dx.x * s_vel_x[k] + dx.y * s_vel_y[k]) / dxSq;	//projection from vel[k] on ortogonal to dx
				float v1_y = dx.y * (dx.x * s_vel_x[k] + dx.y * s_vel_y[k]) / dxSq;
				float v2_x = dx.x * (dx.x * s_vel_x[j] + dx.y * s_vel_y[j]) / dxSq;
				float v2_y = dx.y * (dx.x * s_vel_x[j] + dx.y * s_vel_y[j]) / dxSq;
				float2 dvel = {v1_x - v2_x, v1_y - v2_y};
				dv.x += dvel.x * visc / absDx;
				dv.y += dvel.y * visc / absDx;
			}
		}
		float dvSq = dv.x*dv.x + dv.y*dv.y;
		if (dvSq > max*max)		//max speed
		{
			float c = max / dvSq;
			c = std::sqrt(c);
			dv.x *= c;
			dv.y *= c;
		}
		velN_x[id[k]] = s_vel_x[k] +(dv.x * dt);
		velN_y[id[k]] = s_vel_y[k] + (dv.y * dt);
		posN_x[id[k]] = s_pos_x[k] + (s_vel_x[k] * dt);
		posN_y[id[k]] = s_pos_y[k] + (s_vel_x[k] * dt);
	}
}
hipError_t fluidSimulation(const int2 res, const int2 fields, const float r, const float dt, const float visc, const float d, const float g, const int frames, float* pos_x, float* pos_y);
int main()
{
	const int2 res = { 1000, 800 };
	const int2 fields = {5, 4};
	const float r = 1.f;
	const float dt = 0.01f;
	const float visc = 0.2f;
	const float d = 0.5f;
	const float g = 9.8f;
	const int frames = 100;
	float pos_x[N];
	float pos_y[N];
	float x = 0.f;
	float y = 0.f;
	for (int i = 0; i < N; ++i)
	{
		pos_x[i] = x + (i % 3 == 0 ? 0.5f*r : 0);
		pos_y[i] = y;
		x += 1.8f*r;
		if (x >= res.x)
		{
			x = 0.1f;
			y += 1.8f;
		}
	}
	fluidSimulation(res, fields, r, dt, visc, d, g, frames, pos_x, pos_y);
    return 0;
}

void safeFrame(int num, float* picture, float* dev_picture, const int2 res)	//very slow
{
	hipError_t cudaStatus = hipMemcpy(picture, dev_picture, res.x * res.y * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "copy frame from device failed!" << std::endl;
	}
	copyDevDesFin = false;
	FILE *output;
	std::stringstream ss;
	ss << "frame" << num << ".pnm";
	output = fopen(ss.str().c_str(), "w");
	ss.str("");
	ss << "P5 " << res.x << ' ' << res.y << " 255 ";
	fprintf(output, ss.str().c_str());
	for (int i = 0; i < res.x*res.y; ++i)
	{
		if (picture[i] < 0.f)
		{
			std::cerr << "ERROR" << std::endl;
			return;
		}
		fprintf(output, "%c", picture[i] == 0 ? (int)0 : (int)255);
	}
	fclose(output);
}

hipError_t fluidSimulation(const int2 res, const int2 fields, const float r, const float dt, const float visc, const float d, const float g, const int frames, float* pos_x, float* pos_y)
{
	const float H = 2.f * r;
	const int pixel = res.x * res.y;
	float *dev_vel_x;
	float *dev_vel_y;
	float *dev_pos_x;
	float *dev_pos_y;
	float *dev_velN_x;
	float *dev_velN_y;
	float *dev_posN_x;
	float *dev_posN_y;
	uint32_t *dev_eInG;
	uint4 *dev_areal;
	uint8_t *dev_pic;

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (0 == deviceCount) {
		std::cerr << "No CUDA device found." << std::endl;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDeviceProps(devProp);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
	}
	hipMalloc((void**)dev_vel_x, N * sizeof(float));
	hipMalloc((void**)dev_vel_y, N * sizeof(float));
	hipMalloc((void**)dev_pos_x, N * sizeof(float));
	hipMalloc((void**)dev_pos_y, N * sizeof(float));
	hipMalloc((void**)dev_velN_x, N * sizeof(float));
	hipMalloc((void**)dev_velN_y, N * sizeof(float));
	hipMalloc((void**)dev_posN_x, N * sizeof(float));
	hipMalloc((void**)dev_posN_y, N * sizeof(float));
	hipMalloc((void**)dev_pic, pixel * sizeof(uint8_t));
	hipMalloc((void**)dev_areal, AMOUNT_SM * sizeof(uint4));

	hipMemcpy(dev_pos_x, pos_x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_pos_y, pos_y, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dev_vel_x, 0, N * sizeof(float));
	hipMemset(dev_vel_y, 0, N * sizeof(float));
	hipMemset(dev_pic, 0, N * sizeof(float));

	hipMemcpy(dev_pos_x, pos_x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_pos_y, pos_y, N * sizeof(float), hipMemcpyHostToDevice);

	const int MAX_THREADS_PER_BLOCK = 1024;	//per dim
	
	float* picture = (float*)malloc(pixel * sizeof(float));
	dim3 blockSize = dim3(5, 2);
	dim3 threadSize = dim3(1024);


	std::thread safePicThread;
	const int STEPS_BETWEEN_FRAMES = 30;
	float *dev_p, *dev_diff;		//field to save vel diff and presuare temp
	for (size_t frame = 0; frame <= frames * STEPS_BETWEEN_FRAMES; ++frame)
	{
		//std::cout << "strat " << frame << std::endl;
#ifdef DRAW_PIC
		if (frame % STEPS_BETWEEN_FRAMES == 0)
		{
			if (safePicThread.joinable())
			{
				safePicThread.join();
				copyDevDesFin = true;
				safePicThread = std::thread(safeFrame, frame, picture, dev_pic, res);
				while (copyDevDesFin);
			}
			else
				__debugbreak();
		}
#endif
		//calculate 1 frame
		sumbissionKernel << <MAX_THREADS_PER_BLOCK, 1 >> >
			(res, fields, dev_areal, dev_eInG, H, dt, visc, d, g,
				dev_pos_x, dev_pos_y, dev_vel_x, dev_vel_y, dev_posN_x, dev_posN_y, dev_velN_x, dev_velN_y);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "diffuseKernel failed!" << std::endl;
		}
		std::swap(dev_posN_x, dev_pos_x);
		std::swap(dev_posN_y, dev_pos_y);
		std::swap(dev_velN_x, dev_vel_x);
		std::swap(dev_velN_y, dev_vel_y);
	}
	safePicThread.join();
	hipFree(dev_vel_x);
	hipFree(dev_vel_y);
	hipFree(dev_pos_x);
	hipFree(dev_pos_y);
	hipFree(dev_velN_x);
	hipFree(dev_velN_y);
	hipFree(dev_posN_x);
	hipFree(dev_posN_y);
	hipFree(dev_areal);
	hipFree(dev_pic);

	return cudaStatus;
}