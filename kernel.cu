#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


int main()
{
	const int res = 500;		//image size, resolution per axis
	const float diff = 0.2;	//diffusion speed
	const float dt = 0.1;	//virtual time between to frames
	const int frames = 100;	//amount of frames to render
	//const float src = 1;	//denisty in source field;	TODO:Change to field with production speed per tile
	
	hipError_t cudaStatus = fluidSimulation(res, diff, dt, frames);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames)
{
	float *dev_des;		//field on Device with informatioon about density
	float *dev_des_fin;	//last Completed rendert density field
	float2 *dev_vel;	//field with velocity information		TODO:aproximate Velocity and decress the resolution

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
		goto End;
	}

	size_t pixel = res * res;
	cudaStatus = hipMalloc((void**)&dev_des, pixel * sizeof(float));
	if(cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_des_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel, pixel * sizeof(float2));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed!" << std::endl;
		goto End;
	}
	//TODO:Generate Velocity field
	cudaStatus = hipMemset(dev_des, 0, pixel * sizeof(float));
	if(cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_des_fin, 0, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel, 0, pixel * sizeof(float2));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemset failed!" << std::endl;
		goto End;
	}

End:
	//free all
}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
