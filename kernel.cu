#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <atomic>
#include <thread>

#include "d:\Dokumente\OVGU\GPU\cudaSample\solution\src\cuda_util.h"

hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames);
std::atomic_bool copyDevDesFin(false);
__global__ void addSrcKernel(float* vel_x, float* vel_y, float* des, float* vel_src_x, float* vel_src_y, float* des_src, const float dt, const int res)
{
	int2 id;
	id.x= blockIdx.x * blockDim.x + threadIdx.x;
	id.y= blockIdx.y * blockDim.y + threadIdx.y;
	int i = id.x + id.y * res;
	if (i >= res*res)
		return;

	vel_x[i] += dt * vel_src_x[i];
	if (vel_x[i] > 1.f)
		vel_x[i] = 1.f;
	if (vel_x[i] < -1.f)
		vel_x[i] = -1.f;

	vel_y[i] += dt * vel_src_y[i];
	if (vel_y[i] > 1.f)
		vel_y[i] = 1.f;
	if (vel_y[i] < -1.f)
		vel_y[i] = -1.f;
	
	des[i] += dt * des_src[i];
	if (des[i] > 1.f)
		des[i] = 1.f;
	if (des[i] < 0.f)
		des[i] = 0.f;
}
__global__ void velDiffKernel(float* vel_x, float* vel_y, float* diff,const int res)
{
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	if (id.x >= res || id.y >= res)
		return;
	int i = id.x + id.y * res;

	diff[i] = -0.5f * (std::abs(vel_x[i - 1] - vel_x[i + 1]) + std::abs(vel_y[i - res] - vel_y[i + res]));
}
__global__ void presuerKernel(float* diff, float* p, const int res)
{
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	if (id.x >= res || id.y >= res)
		return;
	int i = id.x + id.y * res;

	p[i] = (diff[i] + diff[i + 1] + diff[i - 1] + diff[i + res] + diff[i - res]) / 5.f;
}
__global__ void pressVelKernel(float* vel_x, float*  vel_y, float* p, const int res, const float dt)
{
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	if (id.x >= res || id.y >= res)
		return;
	int i = id.x + id.y * res;
	vel_x[i] += (p[i + 1] - p[i - 1]) * dt;		//IMP
	if (vel_x[i] > 1.f)
		vel_x[i] = 1.f;
	if (vel_x[i] < -1.f)
		vel_x[i] = -1.f;

	vel_y[i] += (p[i + res] - p[i - res]) * dt;
	if (vel_y[i] > 1.f)
		vel_y[i] = 1.f;
	if (vel_y[i] < -1.f)
		vel_y[i] = -1.f;
}
enum POSITION {TOP, LEFT, RIGHT, BOTTOM};
//TODO pos , dt and one vel are enoug informations
__device__ float denistyLag(POSITION pos, float vel_x, float vel_y, const float dt)
{
	switch (pos)
	{
	case TOP:
		if (vel_y < 0.f)
			return vel_y * dt;
		break;
	case LEFT:
		if (vel_x < 0.f)
			return vel_x * dt;
		break;
	case RIGHT:
		if (vel_x > 0.f)
			return vel_x * dt;
		break;
	case BOTTOM:
		if (vel_y > 0.f)
			return vel_y * dt;
		break;
	}
	return 0.f;
}
__global__ void advectKernel(float *des, float* des_fin, float* vel_x, float* vel_y, const int res, const float dt)	//berchnen wie viel prozent von welcher Zelle nach dt in der aktuellen Zelle landet
{
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	if (id.x >= res || id.y >= res)
		return;
	
	float2 d;		//travelle way from the now center Particel
	d.x = - dt * vel_x[id.y * res + id.x];	//det * vel < 0.5 !!
	d.y = -dt * vel_y[id.y * res + id.x];

	int dx = 1, dy = 1;
	if (d.x < 0.f)
	{
		dx = -1;
		d.x = -d.x;
	}
	if (d.y < 0.f)
	{
		dy = -1;
		d.y = -d.y;
	}
	if (id.x == 0 || id.x == res - 1 || id.y == 0 || id.y == res - 1)	//boundarey, no fluid leg
	{
		des[id.x + id.y * res] = des_fin[id.x + id.y * res];
		if ((id.x != 0 || dx != -1) && (id.x != res - 1 || dx != 1))
			des[id.x + id.y * res] += des_fin[id.x + dx + id.y * res] * d.x * (1.f - d.y);
		else
			des[id.x + id.y * res] -= des_fin[id.x - dx + id.y * res] * denistyLag(dx == -1 ? POSITION::RIGHT : POSITION::LEFT, vel_x[id.x - dx + id.y * res], vel_y[id.x - dx + id.y * res], dt);

		if ((id.y != 0 || dy != -1) && (id.y != res - 1 || dy != 1))
			des[id.x + id.y * res] += des_fin[id.x + (id.y + dy) * res] * (1.f - d.x) * d.y;
		else
			des[id.x + id.y * res] -= des_fin[id.x + (id.y - dy) * res] * denistyLag(dy == -1 ? POSITION::TOP : POSITION::BOTTOM, vel_x[id.x + (id.y - dy) * res], vel_y[id.x + (id.y - dy) * res], dt);

		if ((id.y != 0 || dy != -1) && (id.y != res - 1 || dy != 1) && (id.x != 0 || dx != -1) && (id.x != res - 1 || dx != 1))
			des[id.x + id.y * res] += des_fin[id.x + dx + (id.y + dy) * res] * d.x * d.y;
	}
	else
	{
		des[id.x + id.y * res] = des_fin[id.x + id.y * res] * (1.f - d.x) * (1.f - d.y)
			+ des_fin[id.x + dx + id.y * res] * d.x * (1.f - d.y)
			+ des_fin[id.x + (id.y + dy) * res] * (1.f - d.x) * d.y
			+ des_fin[id.x + dx + (id.y + dy) * res] * d.x * d.y;
	}
}
__global__ void diffuseKernel(float *des, float *des_fin, const int res, const float diff, const float dt)	//diffusion l�sst sich mit einem Faltungsfilter simulieren
{
	//ermittlung der Position
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	//float subtraction = Eviel IMP
	if (id.x >= res || id.y >= res)
		return;
	float sum = 0.f;
	if (id.x > 0)
	{
		sum += des_fin[id.y * res + id.x - 1] - des_fin[id.y * res + id.x];
	}
	if (id.x < res - 1)
	{
		sum += des_fin[id.y * res + id.x + 1] - des_fin[id.y * res + id.x];
	}
	if (id.y > 0)
	{
		sum += des_fin[(id.y - 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	if (id.y < res - 1)
	{
		sum += des_fin[(id.y + 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	des[id.y * res + id.x] = des_fin[id.y * res + id.x] + sum * diff * dt;
	//if (des[id.y * res + id.x] < 0.f)
	//	des[id.y * res + id.x] = 0.f;
	if (des[id.y * res + id.x] > 1.f)
		des[id.y * res + id.x] = 1.f;
}

int main()
{
	const int res = 150;		//image size, resolution per axis
	const float diff = 0.4f;	//diffusion speed
	const float visc = 0.7f;	//viscosity 
	const float dt = 0.3;	//virtual time between to frames
	const int frames = 50;	//amount of frames to render
	//const float src = 1;	//denisty in source field;	TODO:Change to field with production speed per tile
	
	hipError_t cudaStatus = fluidSimulation(res, diff, dt, frames);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
void safeFrame(int num, float* picture, float* dev_picture, const int res)	//very slow
{
	hipError_t cudaStatus = hipMemcpy(picture, dev_picture, res * res * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "copy frame from device failed!" << std::endl;
	}
	copyDevDesFin = false;
	FILE *output;
	std::stringstream ss;
	ss << "frame" << num << ".pnm";
	output = fopen(ss.str().c_str(), "w");
	ss.str("");
	ss << "P5 " << res << ' ' << res << " 255 ";
	fprintf(output, ss.str().c_str());
	for (int i = 0; i < res*res; ++i)
	{
		if (picture[i] < 0.f)
		{
			std::cerr << "ERROR" << std::endl;
			return;
		}
		fprintf(output, "%c", ((int)(picture[i] * 255.f) == '\n' ? (int)(picture[i] * 255.f) + 1 : (int)(picture[i] * 255.f)));
	}
	fclose(output);
}

hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames)
{
	size_t pixel = res * res;

	float *dev_des;		//field on Device with informatioon about density
	float *dev_des_fin;	//last Completed rendert density field
	float *dev_vel_x;	//field with velocity information		TODO:aproximate Velocity and decress the resolution
	float *dev_vel_x_fin;	//use float insted of float2 to reuse diffuseKernel
	float *dev_vel_y;
	float *dev_vel_y_fin;
	float *des_start;	//denisty distribution at start
	float *dev_des_src;	//particel sources
	float *des_src;
	float *dev_vel_src_x;	//velocity sources
	float *dev_vel_src_y;
	float *vel_src_x;
	float *vel_src_y;
	vel_src_x = (float*)malloc(pixel * sizeof(float));
	vel_src_y = (float*)malloc(pixel * sizeof(float));
	int center = res / 2;
	for(size_t j = 0; j < res; ++j)
		for (size_t i = 0; i < res; ++i)
		{
			if (std::abs((int)i - center) < 20 && std::abs((int)j - center) < 20)
				vel_src_y[j * res + i] = -1.f;
			else
				vel_src_y[j * res + i] = 0.f;
			vel_src_x[j * res + i] = 0.f;
		}
	
	des_start = (float*)malloc(pixel * sizeof(float));
	int min = (res / 15)*7;
	int max = (res / 15)*8;
	std::cout << "border for img " << min << " " << max << std::endl;
	for (size_t j = 0; j < res; ++j)
		for (size_t i = 0; i < res; ++i)
		{
			if (i >= min && i <= max && j >= min && j <= max)
				des_start[i + j * res] = .6f;
			else
				des_start[i + j * res] = 0.f;

		}

	des_src = des_start;

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (0 == deviceCount) {
		std::cerr << "No CUDA device found." << std::endl;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDeviceProps(devProp);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
	}

	cudaStatus = hipMalloc((void**)&dev_des, pixel * sizeof(float));
	if(cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_des_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_x, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_y, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_x_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_y_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_src_x, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_src_y, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_des_src, pixel * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed!" << std::endl;
	}
	//TODO:Generate Velocity field
	cudaStatus = hipMemcpy(dev_des_fin, des_start, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_vel_src_x, vel_src_x, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_vel_src_y, vel_src_y, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel_x_fin, 0, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel_y_fin, 0, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_des_src, des_src, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "initialisation failed!" << std::endl;
	}
	const int MAX_THREADS_PER_BLOCK = 32;	//per dim
	unsigned int blocks, threadsPerBlock;
	threadsPerBlock = std::min((int)res, MAX_THREADS_PER_BLOCK);
	blocks = res / MAX_THREADS_PER_BLOCK;
	if (res % MAX_THREADS_PER_BLOCK != 0)
		blocks++;
	std::cout << "need " << blocks << " per dim blocks with max " << threadsPerBlock << "per block per dim" << std::endl;
	float* picture = (float*)malloc(pixel * sizeof(float));
	dim3 blockSize = dim3(blocks, blocks);
	dim3 threadSize = dim3(threadsPerBlock, threadsPerBlock);
	std::thread safePicThread;
	const int STEPS_BETWEEN_FRAMES = 10;
	float *dev_p, *dev_diff;		//field to save vel diff and presuare temp
	for (size_t frame = 0; frame <= frames * STEPS_BETWEEN_FRAMES; ++frame)
	{
		std::cout << "strat " << frame << std::endl;
		if (frame % STEPS_BETWEEN_FRAMES == 0)
		{
			if(safePicThread.joinable())
				safePicThread.join();
			copyDevDesFin = true;
			safePicThread = std::thread(safeFrame, frame, picture, dev_des_fin, res);
		}
		diffuseKernel << <blockSize, threadSize >> > (dev_des, dev_des_fin, res, diff, dt);
		diffuseKernel << <blockSize, threadSize >> > (dev_vel_x, dev_vel_x_fin, res, diff, dt);
		diffuseKernel << <blockSize, threadSize >> > (dev_vel_y, dev_vel_y_fin, res, diff, dt);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "diffuseKernel failed!" << std::endl;
		}
		
		while (copyDevDesFin);	//wait for pic generation
		std::swap(dev_des, dev_des_fin);
		std::swap(dev_vel_x, dev_vel_x_fin);
		std::swap(dev_vel_y, dev_vel_y_fin);

		//calculate very aproxed Velocety evolution over time
		//1. Calculate vel diff
		//2. Calculate preser from this
		//3. Change vel						TODO: make more effizient
		dev_p = dev_vel_y;				//dev_vel_y..Buffer field to temp save pressuar
		dev_diff = dev_vel_x;			//dev_vel_x..Buffer Field
		velDiffKernel << <blockSize, threadSize >> > (dev_vel_x_fin, dev_vel_y_fin, dev_diff, res);
		cudaStatus = hipDeviceSynchronize();
		presuerKernel << <blockSize, threadSize >> > (dev_diff, dev_p, res);
		cudaStatus = hipDeviceSynchronize();
		pressVelKernel << <blockSize, threadSize >> > (dev_vel_x_fin, dev_vel_y_fin, dev_p, res, dt);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "presuerCalculation failed!" << std::endl;
		}
		advectKernel << <blockSize, threadSize >> > (dev_des, dev_des_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);
		advectKernel <<<blockSize, threadSize>>> (dev_vel_x, dev_vel_x_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);
		advectKernel <<<blockSize, threadSize>>> (dev_vel_y, dev_vel_y_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "advect Kernel failed!" << std::endl;
		}

		std::swap(dev_des, dev_des_fin);
		std::swap(dev_vel_x, dev_vel_x_fin);
		std::swap(dev_vel_y, dev_vel_y_fin);

		addSrcKernel << <blockSize, threadSize >> > (dev_vel_x_fin, dev_vel_y_fin, dev_des_fin, dev_vel_src_x, dev_vel_src_y, dev_des_src, dt, res);
		cudaStatus = hipDeviceSynchronize();
	}
	safePicThread.join();
	hipFree(dev_des);
	hipFree(dev_des_fin);
	hipFree(dev_vel_x);
	hipFree(dev_vel_y);
	hipFree(dev_vel_x_fin);
	hipFree(dev_vel_y_fin);

	return cudaStatus;
}