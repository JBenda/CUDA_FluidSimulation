#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <atomic>
#include <thread>

#include "d:\Dokumente\OVGU\GPU\cudaSample\solution\src\cuda_util.h"

hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames);
std::atomic_bool copyDevDesFin(false);
__global__ void addVelKernel(float* vel_x, float* vel_y, float* vel_src_x, float* vel_src_y, const float dt, const int res)
{
	int2 id;
	id.x= blockIdx.x * blockDim.x + threadIdx.x;
	id.y= blockIdx.y * blockDim.y + threadIdx.y;
	int i = id.x + id.y * res;
	vel_x[i] += dt * vel_src_x[i];
	if (vel_x[i] > 1.f)
		vel_x[i] = 1.f;
	if (vel_x[i] < -1.f)
		vel_x[i] = -1.f;
	vel_y[i] += dt * vel_src_y[i];
	if (vel_y[i] > 1.f)
		vel_y[i] = 1.f;
	if (vel_y[i] < -1.f)
		vel_y[i] = -1.f;
}
__global__ void advectKernel(float *des, float* des_fin, float* vel_x, float* vel_y, const int res, const float dt)	//berchnen wie viel prozent von welcher Zelle nach dt in der aktuellen Zelle landet
{
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	if (id.x >= res || id.y >= res)
		return;
	
	float2 d;		//travelle way from the now center Particel
	d.x = - dt * vel_x[id.y * res + id.x];	//det * vel < 0.5 !!
	d.y = - dt * vel_y[id.y * res + id.x];
	//boundary check, wall perfect reflaction
	if ((id.x == 0 && d.x < 0.f) || (id.x == res-1 && d.x > 0.f))
		d.x = 0.f;
	if ((id.y == 0 && d.y < 0.f) || (id.y == res - 1 && d.y > 0.f))
		d.y = 0.f;
	int dx = 1, dy = 1;
	if (d.x < 0.f)
	{
		dx = -1;
		d.x = -d.x;
	}
	if (d.y < 0.f)
	{
		dy = -1;
		d.y = -d.y;
	}
	des[id.x + id.y * res] = des_fin[id.x + id.y * res] * (1.f - d.x) * (1.f - d.y)
		+ des_fin[id.x + dx + id.y * res] * d.x * (1.f - d.y)
		+ des_fin[id.x + (id.y + dy) * res] * (1.f - d.x) * d.y
		+ des_fin[id.x + dx + (id.y + dy) * res] * d.x * d.y;
	if (id.x == 0 || id.x == res - 1 || id.y == 0 || id.y == res - 1)
		des[id.x + id.y * res] = des_fin[id.x + id.y * res] * d.x * d.y;
}
__global__ void diffuseKernel(float *des, float *des_fin, const int res, const float diff, const float dt)	//diffusion l�sst sich mit einem Faltungsfilter simulieren
{
	//ermittlung der Position
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	//float subtraction = Eviel IMP
	if (id.x >= res || id.y >= res)
		return;
	float sum = 0.f;
	if (id.x > 0)
	{
		sum += des_fin[id.y * res + id.x - 1] - des_fin[id.y * res + id.x];
	}
	if (id.x < res - 1)
	{
		sum += des_fin[id.y * res + id.x + 1] - des_fin[id.y * res + id.x];
	}
	if (id.y > 0)
	{
		sum += des_fin[(id.y - 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	if (id.y < res - 1)
	{
		sum += des_fin[(id.y + 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	des[id.y * res + id.x] = des_fin[id.y * res + id.x] + sum * diff * dt;
	//if (des[id.y * res + id.x] < 0.f)
	//	des[id.y * res + id.x] = 0.f;
	if (des[id.y * res + id.x] > 1.f)
		des[id.y * res + id.x] = 1.f;
}

int main()
{
	const int res = 150;		//image size, resolution per axis
	const float diff = 0.4f;	//diffusion speed
	const float visc = 0.7f;	//viscosity 
	const float dt = 0.3;	//virtual time between to frames
	const int frames = 20;	//amount of frames to render
	//const float src = 1;	//denisty in source field;	TODO:Change to field with production speed per tile
	
	hipError_t cudaStatus = fluidSimulation(res, diff, dt, frames);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
void safeFrame(int num, float* picture, float* dev_picture, const int res)	//very slow
{
	hipError_t cudaStatus = hipMemcpy(picture, dev_picture, res * res * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "copy frame from device failed!" << std::endl;
	}
	copyDevDesFin = false;
	FILE *output;
	std::stringstream ss;
	ss << "frame" << num << ".pnm";
	output = fopen(ss.str().c_str(), "w");
	ss.str("");
	ss << "P5 " << res << ' ' << res << " 255 ";
	fprintf(output, ss.str().c_str());
	for (int i = 0; i < res*res; ++i)
	{
		if (picture[i] < 0.f)
		{
			std::cerr << "ERROR" << std::endl;
			return;
		}
		fprintf(output, "%c", ((int)(picture[i] * 255.f) == '\n' ? (int)(picture[i] * 255.f) + 1 : (int)(picture[i] * 255.f)));
	}
	fclose(output);
}

hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames)
{
	size_t pixel = res * res;

	float *dev_des;		//field on Device with informatioon about density
	float *dev_des_fin;	//last Completed rendert density field
	float *dev_vel_x;	//field with velocity information		TODO:aproximate Velocity and decress the resolution
	float *dev_vel_x_fin;	//use float insted of float2 to reuse diffuseKernel
	float *dev_vel_y;
	float *dev_vel_y_fin;
	float *des_start;	//denisty distribution at start
	float *dev_des_src;	//particel sources
	float *dev_vel_src_x;	//velocity sources
	float *dev_vel_src_y;
	float *vel_src_x;
	float *vel_src_y;
	vel_src_x = (float*)malloc(pixel * sizeof(float));
	vel_src_y = (float*)malloc(pixel * sizeof(float));
	int center = res / 2;
	for(size_t j = 0; j < res; ++j)
		for (size_t i = 0; i < res; ++i)
		{
			if (std::abs((int)i - center) < 20 && std::abs((int)j - center) < 20)
				vel_src_y[j * res + i] = -1.f;
			else
				vel_src_y[j * res + i] = 0.f;
			vel_src_x[j * res + i] = 0.f;
		}
	des_start = (float*)malloc(pixel * sizeof(float));
	int min = (res / 4)*2;
	int max = (res / 4)*3;
	std::cout << "border for img " << min << " " << max << std::endl;
	for (size_t j = 0; j < res; ++j)
		for (size_t i = 0; i < res; ++i)
		{
			if (i >= min && i <= max && j >= min && j <= max)
				des_start[i + j * res] = 1.f;
			else
				des_start[i + j * res] = 0.f;

		}

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (0 == deviceCount) {
		std::cerr << "No CUDA device found." << std::endl;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDeviceProps(devProp);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
	}

	cudaStatus = hipMalloc((void**)&dev_des, pixel * sizeof(float));
	if(cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_des_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_x, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_y, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_x_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_y_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_src_x, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel_src_y, pixel * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed!" << std::endl;
	}
	//TODO:Generate Velocity field
	cudaStatus = hipMemcpy(dev_des_fin, des_start, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_vel_src_x, vel_src_x, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_vel_src_y, vel_src_y, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel_x_fin, 0, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel_y_fin, 0, pixel * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "initialisation failed!" << std::endl;
	}
	const int MAX_THREADS_PER_BLOCK = 32;	//per dim
	unsigned int blocks, threadsPerBlock;
	threadsPerBlock = std::min((int)res, MAX_THREADS_PER_BLOCK);
	blocks = res / MAX_THREADS_PER_BLOCK;
	if (res % MAX_THREADS_PER_BLOCK != 0)
		blocks++;
	std::cout << "need " << blocks << " per dim blocks with max " << threadsPerBlock << "per block per dim" << std::endl;
	float* picture = (float*)malloc(pixel * sizeof(float));
	dim3 blockSize = dim3(blocks, blocks);
	dim3 threadSize = dim3(threadsPerBlock, threadsPerBlock);
	std::thread safePicThread;
	for (size_t frame = 0; frame < frames * 40; ++frame)
	{
		addVelKernel << <blockSize, threadSize>> > (dev_vel_x_fin, dev_vel_y_fin, dev_vel_src_x, dev_vel_src_y, dt, res);
		cudaStatus = hipDeviceSynchronize();
		
		std::cout << "strat " << frame << std::endl;
		if (frame % 40 == 0)
		{
			if(safePicThread.joinable())
				safePicThread.join();
			copyDevDesFin = true;
			safePicThread = std::thread(safeFrame, frame, picture, dev_des_fin, res);
		}

		advectKernel << <blockSize, threadSize >> > (dev_des, dev_des_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);
		//TODO: more realistic velocity
		advectKernel <<<blockSize, threadSize>>> (dev_vel_x, dev_vel_x_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);
		advectKernel <<<blockSize, threadSize>>> (dev_vel_y, dev_vel_y_fin, dev_vel_x_fin, dev_vel_y_fin, res, dt);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "advect Kernel failed!" << std::endl;
		}
		std::swap(dev_des, dev_des_fin);
		std::swap(dev_vel_x, dev_vel_x_fin);
		std::swap(dev_vel_y, dev_vel_y_fin);

		diffuseKernel << <blockSize, threadSize >> > (dev_des, dev_des_fin, res, diff, dt);
		diffuseKernel << <blockSize, threadSize >> > (dev_vel_x, dev_vel_x_fin, res, diff, dt);
		diffuseKernel << <blockSize, threadSize >> > (dev_vel_y, dev_vel_y_fin, res, diff, dt);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "diffuseKernel failed!" << std::endl;
		}

		while (copyDevDesFin);
		std::swap(dev_des, dev_des_fin);
		std::swap(dev_vel_x, dev_vel_x_fin);
		std::swap(dev_vel_y, dev_vel_y_fin);
	}
	safePicThread.join();
	hipFree(dev_des);
	hipFree(dev_des_fin);
	hipFree(dev_vel_x);
	hipFree(dev_vel_y);
	hipFree(dev_vel_x_fin);
	hipFree(dev_vel_y_fin);

	return cudaStatus;
}