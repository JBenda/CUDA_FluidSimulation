#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <sstream>

#include "d:\Dokumente\OVGU\GPU\cudaSample\solution\src\cuda_util.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void diffuseKernel(float *des, float *des_fin, const int res, const float diff, const float dt)	//diffusion l�sst sich mit einem Faltungsfilter simulieren
{
	//ermittlung der Position
	int2 id;
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;

	if (id.x >= res || id.y >= res)
		return;
	float sum = 0.f;
	if (id.x > 0)
	{
		sum += des_fin[id.y * res + id.x - 1] - des_fin[id.y * res + id.x];
	}
	if (id.x < res - 1)
	{
		sum += des_fin[id.y * res + id.x + 1] - des_fin[id.y * res + id.x];
	}
	if (id.y > 0)
	{
		sum += des_fin[(id.y - 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	if (id.y < res - 1)
	{
		sum += des_fin[(id.y + 1) * res + id.x] - des_fin[id.y * res + id.x];
	}
	des[id.y * res + id.x] = des_fin[id.y * res + id.x] + sum * diff * dt;
	if (des[id.y * res + id.x] < 0.f)
		des[id.y * res + id.x] = 0.f;
}

int main()
{
	const int res = 100;		//image size, resolution per axis
	const float diff = 0.4f;	//diffusion speed
	const float dt = 0.5;	//virtual time between to frames
	const int frames = 100;	//amount of frames to render
	//const float src = 1;	//denisty in source field;	TODO:Change to field with production speed per tile
	
	hipError_t cudaStatus = fluidSimulation(res, diff, dt, frames);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
void safeFrame(int num, float* picture, const int res)
{
	FILE *output;
	std::stringstream ss;
	ss << "frame" << num << ".pnm";
	output = fopen(ss.str().c_str(), "w");
	ss.str("");
	ss << "P5 " << res << ' ' << res << " 255 ";
	fprintf(output, ss.str().c_str());
	char c;
	for (int i = 0; i < res*res; ++i)
	{
		if (picture[i] < 0.f)
		{
			std::cerr << "ERROR" << std::endl;
			return;
		}
		fprintf(output, "%c", ((int)(picture[i] * 255.f) == '\n' ? (int)(picture[i] * 255.f) + 1 : (int)(picture[i] * 255.f)));
	}
	fclose(output);
}

hipError_t fluidSimulation(const int res, const float diff, const float dt, const int frames)
{
	size_t pixel = res * res;

	float *dev_des;		//field on Device with informatioon about density
	float *dev_des_fin;	//last Completed rendert density field
	float2 *dev_vel;	//field with velocity information		TODO:aproximate Velocity and decress the resolution

	float *des_start;	//denisty distribution at start
	des_start = (float*)malloc(pixel * sizeof(float));
	int min = (res / 3);
	int max = (res / 3)*2;
	std::cout << "border for img " << min << " " << max << std::endl;
	for (size_t i = 0; i < res; ++i)
		for (size_t j = 0; j < res; ++j)
		{
			if (i >= min && i <= max && j >= min && j <= max)
				des_start[i + j * res] = 1.f;
			else
				des_start[i + j * res] = 0.f;
		}

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (0 == deviceCount) {
		std::cerr << "No CUDA device found." << std::endl;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDeviceProps(devProp);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cerr << "hipSetDevice failed!" << std::endl;
		goto End;
	}

	cudaStatus = hipMalloc((void**)&dev_des, pixel * sizeof(float));
	if(cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_des_fin, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMalloc((void**)&dev_vel, pixel * sizeof(float2));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed!" << std::endl;
		goto End;
	}
	//TODO:Generate Velocity field
	cudaStatus = hipMemset(dev_des, 0.f, pixel * sizeof(float));
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemcpy(dev_des_fin, des_start, pixel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus == hipSuccess)
		cudaStatus = hipMemset(dev_vel, 0.f, pixel * sizeof(float2));
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "initialisation failed!" << std::endl;
		goto End;
	}
	const int MAX_THREADS_PER_BLOCK = 32;	//per dim
	size_t blocks, threadsPerBlock;
	threadsPerBlock = std::min((int)res, MAX_THREADS_PER_BLOCK);
	blocks = res / MAX_THREADS_PER_BLOCK;
	if (res % MAX_THREADS_PER_BLOCK != 0)
		blocks++;
	std::cout << "need " << blocks << " per dim blocks with max " << threadsPerBlock << "per block per dim" << std::endl;
	float* picture = (float*)malloc(pixel * sizeof(float));
	dim3 blockSize = dim3(blocks, blocks);
	dim3 threadSize = dim3(threadsPerBlock, threadsPerBlock);
	for (size_t frame = 0; frame < frames * 20; ++frame)
	{
		std::cout << "strat" << std::endl;
		diffuseKernel <<<blockSize, threadSize>>> (dev_des, dev_des_fin, res, diff, dt);
		if (frame % 10 == 0)
		{
			cudaStatus = hipMemcpy(picture, dev_des_fin, pixel * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess)
			{
				std::cerr << "copy frame from device failed!" << std::endl;
			}
			safeFrame(frame, picture, res);
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "diffuseKernel failed!" << std::endl;
			goto End;
		}
		std::swap(dev_des, dev_des_fin);
	}

End:
	hipFree(dev_des);
	hipFree(dev_des_fin);
	hipFree(dev_vel);

	return cudaStatus;
}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
